#include "hip/hip_runtime.h"
//
// Created by egi on 11/3/19.
//

#include "csr_adaptive_spmv.h"
#include "reduce.cuh"

#define NNZ_PER_WG 128u ///< Should be equal to warpSize

template <typename data_type>
__global__ void fill_vector (unsigned int n, data_type *vec, data_type value)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    vec[i] = value;
}

template <typename data_type>
__global__ void csr_adaptive_spmv_kernel (
    const unsigned int n_rows,
    const unsigned int *col_ids,
    const unsigned int *row_ptr,
    const unsigned int *row_blocks,
    const data_type *data,
    const data_type *x,
    data_type *y)
{
  const unsigned int block_row_begin = row_blocks[blockIdx.x];
  const unsigned int block_row_end = row_blocks[blockIdx.x + 1];
  const unsigned int nnz = row_ptr[block_row_end] - row_ptr[block_row_begin];

  __shared__ data_type cache[NNZ_PER_WG];

  if (block_row_end - block_row_begin > 1)
  {
    /// CSR-Stream case
    const unsigned int i = threadIdx.x;
    const unsigned int block_data_begin = row_ptr[block_row_begin];
    const unsigned int thread_data_begin = block_data_begin + i;

    if (i < nnz)
      cache[i] = data[thread_data_begin] * x[col_ids[thread_data_begin]];
    __syncthreads ();

    if ((block_row_begin + i) < block_row_end)
    {
      data_type dot = 0.0;

      // TODO Implement reduce
      for (unsigned int j = row_ptr[block_row_begin + i] - block_data_begin;
           j < row_ptr[block_row_begin + i + 1] - block_data_begin;
           j++)
      {
        dot += cache[j];
      }

      y[block_row_begin + i] = dot;
    }
  }
  else
  {
    const unsigned int row = block_row_begin;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane = threadIdx.x % 32;

    data_type dot = 0;

    if (nnz <= 32)
    {
      /// CSR-Vector case
      if (row < n_rows)
      {
        const unsigned int row_start = row_ptr[row];
        const unsigned int row_end = row_ptr[row + 1];
        unsigned int element = row_start + lane;

        if (element < row_end)
          dot = data[element] * x[col_ids[element]];
      }

      dot = warp_reduce (dot);

      if (lane == 0 && warp_id == 0 && row < n_rows)
        y[row] = dot;
    }
    else
    {
      /// CSR-VectorL case
      if (row < n_rows)
      {
        const unsigned int row_start = row_ptr[row];
        const unsigned int row_end = row_ptr[row + 1];

        for (unsigned int element = row_start + threadIdx.x; element < row_end; element += blockDim.x)
          dot += data[element] * x[col_ids[element]];
      }

      dot = warp_reduce (dot);

      if (lane == 0)
        cache[warp_id] = dot;
      __syncthreads ();

      if (warp_id == 0)
      {
        dot = 0.0;

        for (unsigned int element = lane; element < blockDim.x / 32; element += 32)
          dot += cache[element];

        dot = warp_reduce (dot);

        if (lane == 0 && row < n_rows)
          y[row] = dot;
      }
    }
  }
}

unsigned int
fill_row_blocks (
    unsigned int rows_count,
    const unsigned int *row_ptr,
    unsigned int *row_blocks
)
{
  row_blocks[0] = 0;

  int last_i = 0;
  int current_wg = 1;
  unsigned int nnz_sum = 0;
  for (int i = 1; i <= rows_count; i++)
  {
    nnz_sum += row_ptr[i] - row_ptr[i - 1];

    if (nnz_sum == NNZ_PER_WG)
    {
      last_i = i;
      row_blocks[current_wg++] = i;
      nnz_sum = 0;
    }
    else if (nnz_sum > NNZ_PER_WG)
    {
      if (i - last_i > 1)
      {
        row_blocks[current_wg++] = i - 1;
        i--;
      }
      else
      {
        row_blocks[current_wg++] = i;
      }

      last_i = i;
      nnz_sum = 0;
    }
  }

  row_blocks[current_wg] = rows_count;

  return current_wg;
}

template <typename data_type>
measurement_class gpu_csr_adaptive_spmv (
    const csr_matrix_class<data_type> &matrix,
    resizable_gpu_memory<data_type> &A,
    resizable_gpu_memory<unsigned int> &col_ids,
    resizable_gpu_memory<unsigned int> &row_ptr,
    resizable_gpu_memory<data_type> &x,
    resizable_gpu_memory<data_type> &y,

    data_type*reusable_vector,
    const data_type*reference_y)
{
  auto &meta = matrix.meta;

  const size_t A_size = matrix.get_matrix_size ();
  const size_t col_ids_size = matrix.meta.non_zero_count;
  const size_t row_ptr_size = matrix.meta.rows_count + 1;
  const size_t x_size = matrix.meta.cols_count;
  const size_t y_size = matrix.meta.rows_count;

  A.resize (A_size);
  col_ids.resize (col_ids_size);
  row_ptr.resize (row_ptr_size);
  x.resize (x_size);
  y.resize (y_size);

  hipMemcpy (A.get (), matrix.data.get (), A_size * sizeof (data_type), hipMemcpyHostToDevice);
  hipMemcpy (col_ids.get (), matrix.columns.get (), col_ids_size * sizeof (unsigned int), hipMemcpyHostToDevice);
  hipMemcpy (row_ptr.get (), matrix.row_ptr.get (), row_ptr_size * sizeof (unsigned int), hipMemcpyHostToDevice);

  {
    dim3 block_size = dim3 (512);
    dim3 grid_size {};

    grid_size.x = (x_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (x_size, x.get (), 1.0);

    grid_size.y = (y_size + block_size.x - 1) / block_size.x;
    fill_vector<data_type><<<grid_size, block_size>>> (y_size, y.get (), 0.0);
  }

  // fill delimiters
  std::unique_ptr<unsigned int[]> row_blocks(new unsigned int[meta.rows_count + 1]);

  const unsigned int blocks_count = fill_row_blocks (meta.rows_count, matrix.row_ptr.get (), row_blocks.get ());

  unsigned int *d_row_blocks {};
  hipMalloc (&d_row_blocks, (meta.rows_count + 1) * sizeof (unsigned int));
  hipMemcpy (d_row_blocks, row_blocks.get (), sizeof (unsigned int) * (meta.rows_count + 1), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

  hipDeviceSynchronize ();
  hipEventRecord (start);
  {
    dim3 block_size = dim3 (NNZ_PER_WG);
    dim3 grid_size {};

    grid_size.x = blocks_count; // (meta.non_zero_count + block_size.x - 1) / block_size.x;

    csr_adaptive_spmv_kernel<<<grid_size, block_size>>> (
        meta.rows_count, col_ids.get (), row_ptr.get (), d_row_blocks, A.get (), x.get (), y.get ());
  }
  hipEventRecord (stop);
  hipEventSynchronize (stop);

  float milliseconds = 0;
  hipEventElapsedTime (&milliseconds, start, stop);

  hipMemcpy (reusable_vector, y.get (), y_size * sizeof (data_type), hipMemcpyDeviceToHost);
  hipFree (d_row_blocks);

  compare_results (y_size, reusable_vector, reference_y);

  const double elapsed = milliseconds / 1000;

  const size_t data_bytes = matrix.meta.non_zero_count * sizeof (data_type);
  const size_t x_bytes = matrix.meta.non_zero_count * sizeof (data_type);
  const size_t col_ids_bytes = matrix.meta.non_zero_count * sizeof (unsigned int);
  const size_t row_ids_bytes = 2 * matrix.meta.rows_count * sizeof (unsigned int);
  const size_t y_bytes = matrix.meta.rows_count * sizeof (data_type);

  const size_t operations_count = matrix.meta.non_zero_count * 2; // + and * per element

  return measurement_class (
      "GPU CSR-Adaptive",
      elapsed,
      data_bytes + x_bytes + col_ids_bytes + row_ids_bytes + y_bytes,
      operations_count);
}


#define INSTANTIATE(data_type)                                                 \
  template measurement_class gpu_csr_adaptive_spmv<data_type>(                 \
      const csr_matrix_class<data_type> &matrix,                               \
      resizable_gpu_memory<data_type> &A,                                      \
      resizable_gpu_memory<unsigned int> &col_ids,                             \
      resizable_gpu_memory<unsigned int> &row_ptr,                             \
      resizable_gpu_memory<data_type> &x, resizable_gpu_memory<data_type> &y,  \
      data_type *reusable_vector, const data_type *reference_y);


INSTANTIATE (float)
INSTANTIATE (double)
#undef INSTANTIATE
